#include "hip/hip_runtime.h"
/*
*       V2. GPU with one thread computing a block of moments
*       Author:Tsalidis Georgios 5/1/2020
*       gtsalidis@ece.auth.gr
*/

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "../inc/ising.h"

#define BLOCK_SIZE 128 // value usually chosen by tuning and hardware constraints
#define CUDA_CHECK_ERROR() __cuda_check_errors(__FILE__, __LINE__)

// See: http://codeyarns.com/2011/03/02/how-to-do-error-checking-in-cuda/
inline void
__cuda_check_errors (const char *filename, const int line_number)
{
  hipError_t err = hipDeviceSynchronize ();
  if (err != hipSuccess)
    {
      printf ("CUDA error %i at %s:%i: %s\n",
          err, filename, line_number, hipGetErrorString (err));
      exit (-1);
    }
}


__global__ void ising_kernel(double* gpu_w, int* gpu_G, int* gpu_Gtmp, int n);
bool evaluate(int *G1,int *G2, int n);


//kernel function used to calculate one thread with a block of moments
__global__ void ising_kernel(double* gpu_w, int* gpu_G, int* gpu_Gtmp, int n)
{
	//calculate thread_id
	int thread_id = blockIdx.x*blockDim.x + threadIdx.x;

	// the value of each moment
	double influence;

	// moments x,y coordinates
	int x, y;
	 
	// Indexes of neibghors checked
	int idx_x, idx_y;
	int next_thr = gridDim.x*blockDim.x;

	//each thread to compute a block of moments
	for(int thread = thread_id; thread<n*n; thread+= next_thr)
	{
		// moments x,y coordinates
		y = thread%n;
		x = thread/n;
		influence = 0;
		// loop through the moment neighbors
	    for(int X=0; X<5; X++)
	        for(int Y=0; Y<5; Y++)
	        {
				// skips the current iteration of the loop and continues with the next iteration.
	            if((X == 2) && (Y == 2))
	                continue;  
				
	            //find idx of checked point
	            idx_x = (x + (X-2) + n) % n;
	            idx_y = (y + (Y-2) + n) % n;

	            influence += *(gpu_w + X*5 + Y) * *(gpu_G +idx_x*n + idx_y);
	        }

	    //the value of the sign of influence If positive -> 1,If negative -> -1
		if(influence > 0.0001)
		{
			*(gpu_Gtmp + x*n + y) = 1;
		}
		else if(influence < -0.0001)
		{
			*(gpu_Gtmp + x*n + y) = -1;
		}
	    else
			//remains the same
	        *(gpu_Gtmp + x*n + y) = *(gpu_G + x*n + y);
	}
}


void ising(int *G, double *w, int k, int n)
{
   
	double *gpu_w;
	int *gpu_G;
	
	// allocate weight array and G array
	hipMalloc(&gpu_w, 5*5*sizeof(double));
	hipMalloc(&gpu_G, n*n*sizeof(int));
	
	
	//transfer data to device(GPU)
	hipMemcpy(gpu_w, w, 5*5*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(gpu_G, G, n*n*sizeof(int), hipMemcpyHostToDevice);

	//GPU array to store the updated values
	int *gpu_Gtmp;
	hipMalloc(&gpu_Gtmp, n*n*sizeof(int));

	// gpu_G with gpu_Gtmp pointer swap
	int *temp;

	//dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	//dim3 dimGrid(GRID_DIM_X,GRID_DIM_Y);
	int block = BLOCK_SIZE;
	int grid = (n + block -1)/block;

	//run for k iterations
	for(int i = 0; i < k; i++)
	{
		//run kernel function to device
		ising_kernel<<< grid , block >>>(gpu_w, gpu_G, gpu_Gtmp, n);
		
		//check for device errors
		CUDA_CHECK_ERROR ();
		
		//Synchronize 
		hipDeviceSynchronize();

		//swap pointers 
		temp = gpu_G;
		gpu_G = gpu_Gtmp;
		gpu_Gtmp = temp;
	}

	hipMemcpy(G, gpu_G, n*n*sizeof(int), hipMemcpyDeviceToHost);

	// free GPU memory
	hipFree(gpu_w);
	hipFree(gpu_G);
	hipFree(gpu_Gtmp);
}